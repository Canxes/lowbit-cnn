#include "hip/hip_runtime.h"
#include "benchmark.cuh"

void validate_call(const hipError_t& err)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error occurred: " << err << " " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("CUDA error");
    }
}

void validate_call(const hipdnnStatus_t& err)
{
    if (err != HIPDNN_STATUS_SUCCESS)
    {
        std::cerr << "cuDNN error occurred: " << err << " " << hipdnnGetErrorString(err) << std::endl;
        throw std::runtime_error("cuDNN error");
    }
}

void log(int verbose, std::ostream& ostream, std::string str)
{
    validate_call(hipDeviceSynchronize());
    if (verbose)
        ostream << str << std::endl;
}

template <typename T>
__global__ void fill_with_constant(T *px, T k)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    px[tid] = k;
}

bool first = true;
template <typename InputDataType, typename FilterDataType, typename OutDataType>
std::chrono::microseconds benchmark_convolution(size_t B, size_t C, size_t H, size_t W,
                                                size_t numFilters, size_t filterH, size_t filterW,
                                                size_t padH, size_t padW, size_t strideH, size_t strideW, size_t dilationH, size_t dilationW,
                                                hipdnnTensorFormat_t inputTensorFormat, hipdnnTensorFormat_t filterTensorFormat, hipdnnTensorFormat_t outputTensorFormat, 
                                                hipdnnDataType_t inputDataType, hipdnnDataType_t filterDataType, 
                                                hipdnnDataType_t convAccumulatorDataType, hipdnnDataType_t outDataType,
                                                int verbose)
{
    if (first)
    {
        first = false;
        validate_call(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
    }

    hipdnnHandle_t cudnn = nullptr;

    hipdnnTensorDescriptor_t inputDescriptor = nullptr;
    InputDataType *inputData = nullptr;

    hipdnnFilterDescriptor_t filterDescriptor = nullptr;
    FilterDataType *filterData = nullptr;
    hipdnnConvolutionDescriptor_t convDescriptor = nullptr;

    hipdnnTensorDescriptor_t outDescriptor = nullptr;
    OutDataType *outData = nullptr;

    hipdnnConvolutionFwdAlgoPerf_t convAlgo;
    void *workspaceData = nullptr;

    std::chrono::microseconds elapsed;

    try
    {
        // Create cudnn
        validate_call(hipdnnCreate(&cudnn));
        log(verbose, std::clog, "Cudnn created");

        // Create input tensor
        validate_call(hipdnnCreateTensorDescriptor(&inputDescriptor));
        validate_call(hipdnnSetTensor4dDescriptor(inputDescriptor, inputTensorFormat, inputDataType, B, C, H, W));

        validate_call(hipMalloc(&inputData, B * C * H * W * sizeof(InputDataType)));
        log(verbose, std::clog, "Input tensor allocated");

        // Create filter descriptor
        validate_call(hipdnnCreateFilterDescriptor(&filterDescriptor));
        validate_call(hipdnnSetFilter4dDescriptor(filterDescriptor, filterDataType, filterTensorFormat, numFilters, C, filterH, filterW));

        validate_call(hipMalloc(&filterData, numFilters * C * filterH * filterW * sizeof(FilterDataType)));
        log(verbose, std::clog, "Filter tensor allocated");

        // Convolution descriptor
        validate_call(hipdnnCreateConvolutionDescriptor(&convDescriptor));
        validate_call(hipdnnSetConvolution2dDescriptor(convDescriptor, padH, padW, strideH, strideW,
                                                    dilationH, dilationW, HIPDNN_CONVOLUTION, convAccumulatorDataType));
        validate_call(hipdnnSetConvolutionMathType(convDescriptor, HIPDNN_TENSOR_OP_MATH));
        log(verbose, std::clog, "Convolution descriptor created");
        int outB, outC, outH, outW;
        validate_call(hipdnnGetConvolution2dForwardOutputDim(convDescriptor, inputDescriptor, filterDescriptor, &outB, &outC, &outH, &outW));
        log(verbose, std::clog, "Computed convolution output shape");

        // Output tensor
        validate_call(hipdnnCreateTensorDescriptor(&outDescriptor));
        validate_call(hipdnnSetTensor4dDescriptor(outDescriptor, outputTensorFormat, outDataType, outB, outC, outH, outW));
        validate_call(hipMalloc(&outData, outB * outC * outH * outW * sizeof(OutDataType)));
        log(verbose, std::clog, "Output tensor allocated");

        // Algorithm
        int foundAlgo;
        validate_call(hipdnnFindConvolutionForwardAlgorithm(
                cudnn, inputDescriptor, filterDescriptor, convDescriptor, outDescriptor, 1, &foundAlgo, &convAlgo));
        if (foundAlgo == 0 || convAlgo.determinism == CUDNN_NON_DETERMINISTIC || convAlgo.status != HIPDNN_STATUS_SUCCESS)
        {
            log(verbose, std::clog, "Best algorithm is non deterministic or not found. Terminating.");
            throw std::runtime_error("Failed to find cudnn algorithm for convolution.");
        }
        log(verbose, std::clog, "Best algorithm is chosen " + std::to_string(convAlgo.algo) + " with math " + std::to_string(convAlgo.mathType));

        if (convAlgo.mathType == HIPDNN_TENSOR_OP_MATH)
            log(verbose, std::clog, "Using Tensor CORES!!!");

        // Workspace
        size_t workspaceSize = convAlgo.memory;
        if (workspaceSize != 0){}
            validate_call(hipMalloc(&workspaceData, workspaceSize));
        log(verbose, std::clog, "Workspace is allocated");

        // Convolution
        float alpha = 1.0f;
        float beta = 0.0f;

        // Dummy values
        ::fill_with_constant<<<numFilters*filterW * filterH, C>>>(filterData, (FilterDataType)2);
        ::fill_with_constant<<<W * H, B * C>>>(inputData, (InputDataType)1);
        log(verbose, std::clog, "Filled with dummy values");

        validate_call(hipDeviceSynchronize());
        auto begin = std::chrono::high_resolution_clock::now();

        validate_call(hipdnnConvolutionForward(
                cudnn,
                &alpha, inputDescriptor, inputData, filterDescriptor, filterData,
                convDescriptor, convAlgo.algo, workspaceData, workspaceSize,
                &beta, outDescriptor, outData));

        validate_call(hipDeviceSynchronize());
        elapsed = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - begin);
        log(verbose, std::clog, "Finalizing");
    }
    catch(const std::exception& e)
    {
        std::cerr << "Error during convolution forward. Returned value is 0. Releasing resources..." << '\n';
        elapsed = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::seconds(0));
    }
    

    // Finalizing
    if (workspaceData != nullptr)
        validate_call(hipFree(workspaceData));

    if (outData != nullptr)
        validate_call(hipFree(outData));
    if (outDescriptor != nullptr)
        validate_call(hipdnnDestroyTensorDescriptor(outDescriptor));
    log(verbose, std::clog, "Out tensor destroyed");
    
    if (convDescriptor != nullptr)
        validate_call(hipdnnDestroyConvolutionDescriptor(convDescriptor));
    log(verbose, std::clog, "Conv descriptor destroyed");
    
    if (filterData != nullptr)
        validate_call(hipFree(filterData));
    if (filterDescriptor != nullptr)
        validate_call(hipdnnDestroyFilterDescriptor(filterDescriptor));
    log(verbose, std::clog, "Filter tensor destroyed");

    if (inputData != nullptr)
        validate_call(hipFree(inputData));
    if (inputDescriptor != nullptr)
        validate_call(hipdnnDestroyTensorDescriptor(inputDescriptor));
    log(verbose, std::clog, "Input tensor destroyed");

    if (cudnn != nullptr)
        validate_call(hipdnnDestroy(cudnn));
    log(verbose, std::clog, "Cudnn destroyed");
    
    return elapsed;
};


// FLOAT CONFIG
template std::chrono::microseconds benchmark_convolution<float, float, float>(
    size_t, size_t, size_t, size_t, size_t, size_t, size_t,
    size_t, size_t, size_t, size_t, size_t, size_t,
    hipdnnTensorFormat_t, hipdnnTensorFormat_t, hipdnnTensorFormat_t, 
    hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, int);
// TRUE_HALF CONFIG
template std::chrono::microseconds benchmark_convolution<half, half, half>(
    size_t, size_t, size_t, size_t, size_t, size_t, size_t,
    size_t, size_t, size_t, size_t, size_t, size_t,
    hipdnnTensorFormat_t, hipdnnTensorFormat_t, hipdnnTensorFormat_t, 
    hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, int);
// DOUBLE CONFIG
template std::chrono::microseconds benchmark_convolution<double, double, double>(
    size_t, size_t, size_t, size_t, size_t, size_t, size_t,
    size_t, size_t, size_t, size_t, size_t, size_t,
    hipdnnTensorFormat_t, hipdnnTensorFormat_t, hipdnnTensorFormat_t, 
    hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, int);
// INT8* CONFIG
template std::chrono::microseconds benchmark_convolution<int8_t, int8_t, int8_t>(
    size_t, size_t, size_t, size_t, size_t, size_t, size_t,
    size_t, size_t, size_t, size_t, size_t, size_t,
    hipdnnTensorFormat_t, hipdnnTensorFormat_t, hipdnnTensorFormat_t, 
    hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, int);
// INT8*_EXT CONFIG
template std::chrono::microseconds benchmark_convolution<int8_t, int8_t, float>(
    size_t, size_t, size_t, size_t, size_t, size_t, size_t,
    size_t, size_t, size_t, size_t, size_t, size_t,
    hipdnnTensorFormat_t, hipdnnTensorFormat_t, hipdnnTensorFormat_t, 
    hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, int);
// UINT8* CONFIG
template std::chrono::microseconds benchmark_convolution<uint8_t, int8_t, int8_t>(
    size_t, size_t, size_t, size_t, size_t, size_t, size_t,
    size_t, size_t, size_t, size_t, size_t, size_t,
    hipdnnTensorFormat_t, hipdnnTensorFormat_t, hipdnnTensorFormat_t,
    hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, int);
// UINT8*_EXT CONFIG
template std::chrono::microseconds benchmark_convolution<uint8_t, int8_t, float>(
    size_t, size_t, size_t, size_t, size_t, size_t, size_t,
    size_t, size_t, size_t, size_t, size_t, size_t,
    hipdnnTensorFormat_t, hipdnnTensorFormat_t, hipdnnTensorFormat_t, 
    hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, hipdnnDataType_t, int);